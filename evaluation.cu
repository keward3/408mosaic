#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sdtlib.h>
#include <hip/hip_runtime.h>

    /* comporfull should be zero when computing for the component image array
     *   and one when computing for sections of the full image
     * when comporfull is zero, numimages is the number of input component images
     * when comporfull is one, numimages is the number of sections in the full image 
     * numwide is used only for full image computation - holds number of sections
     *   needed to fill one row of the full image - input zero if comporfull is zero
     */
__global__ void histandcompval(unsigned char* imagearray, int* compvals, int comporfull, int numimages, int size, int height, int width, int numwide)
{
    __shared__ unsigned int privhistr[256];
    __shared__ unsigned int privhistg[256];
    __shared__ unsigned int privhistb[256];
    
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int by = blockIdx.y; //4 blocks of 256 threads

    quadrant_offset = 0;
    if(bx == 1){ quadrant_offset += width; }
    if(by == 1){ 
        if(comporfull == 0){ quadrant_offset += (size << 1); }
        else{ quadrant_offset += (numwide * size << 1); }
    }

    for(int j = 0; j < numimages; j++){
    
    privhistr[tx] = 0;
    privhistg[tx] = 0;
    privhistb[tx] = 0;
    __syncthreads();

    if(comporfull == 0){ int start = (j << 2) * size + quadrant_offset; }
    else{
        int start = ((j << 1) * (size << 2) + (j << 1) % numwide) * size + quadrant_offset;
    }
    
    for(int i = 0; i * 256 < size; i++){
        quadrant_x = (tx + i * 256) % width;
        quadrant_y = (tx + i * 256) / width;
        if(comporfull == 1){ width *= (numwide << 1); }
        if(quadrant_y < height){
            atomicAdd(&(privhistr[imagearray[start + (quadrant_x + quadrant_y * width) * 3]]), 1);
            atomicAdd(&(privhistg[imagearray[start + (quadrant_x + quadrant_y * width) * 3 + 1]), 1);
            atomicAdd(&(privhistb[imagearray[start + (quadrant_x + quadrant_y * width) * 3 + 2]), 1);
        }
        __syncthreads();
    }

    privhistr[tx] *= tx;
    privhistg[tx] *= tx;
    privhistb[tx] *= tx;
    __syncthreads();

    for(stride = 128; stride > 0; stride >> 1){
        if(tx < stride){
            privhistr[tx] += privhistr[tx + stride];
            privhistg[tx] += privhistg[tx + stride];
            privhistb[tx] += privhistb[tx + stride];
        }
    __syncthreads();
    }

    if(tx == 0){
        compvals[3 * (bx + (by << 1) + (j << 2))] = privhistr[0];
        compvals[1 + 3 * (bx + (by << 1) + (j << 2))] = privhistg[0];
        compvals[2 + 3 * (bx + (by << 1) + (j << 2))] = privhistb[0];
    }
    __syncthreads();

    }
}


    /* kernel call for component image evaluation */
    int* dev_compvals;
    hipMalloc((void**)&dev_compvals, numsections * sizeof(int));
    int half_compheight = COMP_HEIGHT >> 1;
    int half_compwidth = COMP_WIDTH >> 1;
    int comp_quadrantsize = half_compheight * half_compwidth;
    histandcompval<<<4,256>>>(dev_compimagearray, dev_compcompvals, 0, numcompimages, comp_quadrantsize, half_compheight, half_compwidth, 0)
    //eventually free devcompvals

    /*kernel call for full image evaluation */
    int fullimageheight = 0; //assign a real value
    int fullimagewidth = 0; //assign a real value
    int* dev_sectcompvals;
    hipMalloc((void**)&dev_sectcompvals, numsections * sizeof(int));
    int halfsectionheight = fullimageheight / (FINAL_HEIGHT / COMP_HEIGHT) >> 1;
    int sectionswide = FINAL_WIDTH / COMP_WIDTH;
    int halfsectionwidth = fullimagewidth / sectionswide >> 1;
    int full_quadrantsize = halfsectionheight * halfsectionwidth;
    histandcompval<<<4,256>>>(dev_fullimage, dev_sectcompvals, 1, numsections, full_quadrantsize, halfsectionheight, halfsectionwidth, sectionswide);
    //eventually free sectcompvals
